#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>

#include <opencv2/opencv.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/core.hpp>
#include "VirtualSensor.h"

#include "core/VolumetricFusion.h"
#include "core/raycasting.h"
#include "core/preprocessing.h"
#include "core/declarations.h"
#include "core/frame.h"
#include "core/PoseEstimation.h"

#define CALCULATE_DEPTH_LEVELS 0
#define PLOT_RECONSTRUCTION 0
#define PLOT_DEPTH_LEVELS 0

using namespace std;
using namespace cv;
using namespace cuda;

int grid_size = 256;
float min_x = -2;
float max_x = 2;
float min_y = -2;
float max_y = 2;
float min_z = -2;
float max_z = 2;

float truncation = 0.1f;

float minDepth = 0.1f;
float maxDepth = 3;

int main()
{
    float* W = new float[grid_size*grid_size*grid_size];
    float* F = new float[grid_size*grid_size*grid_size];

    for(int x = 0; x < grid_size; x++) {
        for(int y = 0; y < grid_size; y++) {
            for(int z = 0; z < grid_size; z++) { // initialize the values to whatever you want the default to be
                F[x * grid_size * grid_size + y * grid_size + z] = 1;
                W[x * grid_size * grid_size + y * grid_size + z] = 0;
            }
        }
    }

    // Make sure this path points to the data folder
    std::string filenameIn = "../Data/rgbd_dataset_freiburg1_xyz/";
    std::string filenameBaseOut = "mesh_";

    // load video
    VirtualSensor sensor;
    if (!sensor.init(filenameIn)){
        std::cout << "Failed to initialize the sensor!\nCheck file path!" << std::endl;
        return -1;
    } else {
        std::cout << "File Opened" << std::endl;
    }


    Vertex* vertices_d;
    Normal* normals_d;

    float* phongSurface_d;
    float* phongSurface_curr_d;

    float* F_d;
    float* W_d;
    float* depth_d;

    Vector3f* predictedNormals_d;
    Vector3f* predictedVertices_d;
    Vector3f* predictedNormals_curr_d;

    float num_voxels = grid_size*grid_size*grid_size;
    int numVertices = 640*480;
    int* vertex_validity_d;

    hipMalloc(&normals_d, numVertices*sizeof(Normal));
    hipMalloc(&vertices_d, numVertices*sizeof(Vertex));
    hipMalloc(&vertex_validity_d, numVertices*sizeof(int));

    hipMalloc(&F_d, num_voxels*sizeof(float));

    hipMalloc(&W_d, num_voxels*sizeof(float));
    hipMalloc(&vertex_validity_d, numVertices*sizeof(int));
    hipMalloc(&depth_d, numVertices*sizeof(float));
    hipMalloc(&predictedNormals_d, numVertices*sizeof(Vector3f));
    hipMalloc(&predictedNormals_curr_d, numVertices*sizeof(Vector3f));

    hipMalloc(&predictedVertices_d, numVertices*sizeof(Vector3f));
    hipMalloc(&phongSurface_d, numVertices*sizeof(float));
    hipMalloc(&phongSurface_curr_d, numVertices*sizeof(float));

    hipMemcpy(F_d, F, num_voxels*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(W_d, W, num_voxels*sizeof(float), hipMemcpyHostToDevice);

    Matrix4f previousPose;
    Vector3f* predictedNormals = new Vector3f[numVertices];
    Vector3f* predictedVertices = new Vector3f[numVertices];
    Matrix4f depthExtrinsics = Matrix4f::Identity();
    Matrix4f depthExtrinsicsInv = Matrix4f::Identity();

    Frame frame;
    PoseEstimation cameraPose;

    int sensor_frame = 0;
    while(sensor.processNextFrame()) {
        auto start_total = chrono::steady_clock::now();
        sensor_frame+= 1;
        float* depthMat = sensor.getDepth();
        unsigned int depthWidth = sensor.getDepthImageWidth();
        unsigned int depthHeight = sensor.getDepthImageHeight();

        Matrix3f depthIntrinsics = sensor.getDepthIntrinsics();     // get K matrix (intrinsics), global to camera frame
        Matrix3f depthIntrinsicsInv = depthIntrinsics.inverse();
        Matrix3f intrinsicsInv = depthIntrinsics.inverse();
        Matrix4f depthExtrinsics = sensor.getTrajectory();
        Matrix4f depthExtrinsicsInv = depthExtrinsics.inverse();
        Matrix4f initialPose;

        if(sensor_frame == 1) {
            depthExtrinsics = sensor.getTrajectory();
            depthExtrinsicsInv = depthExtrinsics.inverse();
            previousPose = depthExtrinsics;
            previousPose = Matrix<float,4,4>::Identity();
            initialPose = depthExtrinsics; //for render sdf
//            initialPose = sensor.getTrajectory();
        }

        cv::Mat depth_mat = cv::Mat(static_cast<int>(depthHeight), static_cast<int>(depthWidth), CV_32F, depthMat);
        cv::Mat filt_depth_mat = cv::Mat(static_cast<int>(depthHeight), static_cast<int>(depthWidth), CV_32F);
//        filt_depth_mat = frame.applyBilateral(depth_mat, depthHeight, depthWidth);

//------- GPU bilateral filter implementation------
        auto start_filt = chrono::steady_clock::now();
        GpuMat filt_depth_mat_g;
        GpuMat depth_map_g;
        depth_map_g.upload(depth_mat);
        cv::cuda::bilateralFilter(depth_map_g, filt_depth_mat_g, 9, 9, 0,
                                  BORDER_DEFAULT); //max d val of 5 recommended for real-time applications (9 for offline)
        filt_depth_mat_g.download(filt_depth_mat);
        auto end_filt = chrono::steady_clock::now();

        Vertex* vertices = new Vertex[numVertices];
        Normal* normals = new Normal[numVertices];
        int* vertex_validity = new int[numVertices];

        if (CALCULATE_DEPTH_LEVELS){
            frame.subSampleDepthLevels(filt_depth_mat);
            frame.computeVerticeLevels_cpu(depthIntrinsicsInv);
            frame.computeNormalLevels_cpu();

            vertices = frame.getVertices(1);  //copy original (level1)bra vertices back for cuda vars
            normals  = frame.getNormals(1);
            vertex_validity = frame.getVertexValidity(1);
        }
        else {
//            otherwise, just calculate once the old way
            for (unsigned int r = 0; r < depthHeight; r++) {
                for (unsigned int c = 0; c < depthWidth; c++) {
                    unsigned int vertex_idx = r * depthWidth + c;
                    float depth_pixel = filt_depth_mat.at<float>(r, c);
                    normals[vertex_idx].val = Vector3f(1.0f, 1.0f, 1.0f);

                    if (isnan(depth_pixel)) {
                        vertex_validity[vertex_idx] = 0;
                        vertices[vertex_idx].pos = Vector4f(MINF, MINF, MINF, MINF);
                    } else {
                        Vector3f camera_coord = depthIntrinsicsInv * Vector3f(c, r, 1) * depth_pixel;
                        vertices[vertex_idx].pos[0] = camera_coord[0];
                        vertices[vertex_idx].pos[1] = camera_coord[1];
                        vertices[vertex_idx].pos[2] = camera_coord[2];
                        vertices[vertex_idx].pos[3] = 1.0f;
                        vertex_validity[vertex_idx] = 1;
                    }
                }
            }
        }

        auto start_tsdf = chrono::steady_clock::now();
        dim3 threads(30,30);
        dim3 blocks((depthWidth+29) / 30, (depthHeight+29) / 30);

        hipMemcpy(vertices_d, vertices, numVertices*sizeof(Vertex), hipMemcpyHostToDevice);
        hipMemcpy(vertex_validity_d, vertex_validity, numVertices*sizeof(int), hipMemcpyHostToDevice);

        computeNormals<<<blocks,threads>>>(vertices_d, vertex_validity_d, normals_d, depthWidth, depthHeight);
        hipMemcpy(normals, normals_d, numVertices*sizeof(Normal), hipMemcpyDeviceToHost);

        hipMemcpy(vertex_validity_d, vertex_validity, numVertices*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(depth_d, (float*) filt_depth_mat.data, numVertices*sizeof(float), hipMemcpyHostToDevice);

        if (sensor_frame != 1){
//            Matrix4f currPose = cameraPose.estimatePose(vertices, normals, predictedVertices, predictedNormals, vertex_validity,
//                                           previousPose, depthIntrinsics, depthWidth, depthHeight);
//            cout << "Passing following to func" << endl << previousPose << endl;
//            Matrix4f currPose = cameraPose.estimatePose3(vertices, normals, predictedVertices, predictedNormals, vertex_validity,
//                                                         depthIntrinsics, previousPose, depthHeight,depthWidth);
//            depthExtrinsics = cameraPose.estimatePose(vertices, normals, predictedVertices, predictedNormals, vertex_validity,
//                                            depthIntrinsics,previousPose, depthWidth, depthHeight);
//            cout << "Estimated pose: " << endl << currPose << endl;
//            previousPose = currPose;
//            depthExtrinsics = currPose;
            depthExtrinsics = sensor.getTrajectory();
            depthExtrinsicsInv = depthExtrinsics.inverse();
            delete[] predictedNormals;
            delete[] predictedVertices;
            predictedNormals = new Vector3f[640*480];
            predictedVertices = new Vector3f[640*480];
        }

        Vector3f* predictedNormals = new Vector3f[depthWidth*depthHeight];
        Vector3f* predictedVertices = new Vector3f[depthWidth*depthHeight];
        float* phongSurface = new float[depthWidth*depthHeight];
        float* phongSurface_curr = new float[depthWidth*depthHeight];

//        auto start_tsdf = chrono::steady_clock::now();
        dim3 threads_tsdf(10,10,10);
        dim3 blocks_tsdf((grid_size+9) / 10, (grid_size+9) / 10, (grid_size+9) / 10);
        updateTSDF<<<blocks_tsdf, threads_tsdf>>>(F_d, W_d, depthExtrinsics, depthExtrinsicsInv, depth_d, normals_d, vertex_validity_d, depthWidth, depthHeight, depthIntrinsics, depthIntrinsicsInv, grid_size, truncation, min_x, max_x, min_y, max_y, min_z, max_z);
        hipDeviceSynchronize();
        auto end_tsdf = chrono::steady_clock::now();

        auto start_raycast = chrono::steady_clock::now();
        RenderTSDF<<<blocks,threads>>>(F_d, initialPose, depthIntrinsics,predictedVertices_d, predictedNormals_d, depthWidth, depthHeight, phongSurface_d, grid_size, minDepth, maxDepth, min_x, max_x, min_y, max_y, min_z, max_z);
        hipDeviceSynchronize();

        RenderTSDF<<<blocks,threads>>>(F_d, depthExtrinsics, depthIntrinsics,predictedVertices_d, predictedNormals_curr_d, depthWidth, depthHeight, phongSurface_curr_d, grid_size, minDepth, maxDepth, min_x, max_x, min_y, max_y, min_z, max_z);
        hipDeviceSynchronize();
//        auto end_raycast = chrono::steady_clock::now();

        hipMemcpy(phongSurface, phongSurface_d, numVertices*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(phongSurface_curr, phongSurface_curr_d, numVertices*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(predictedNormals, predictedNormals_d, numVertices*sizeof(Vector3f), hipMemcpyDeviceToHost);
        hipMemcpy(predictedVertices, predictedVertices_d, numVertices*sizeof(Vector3f), hipMemcpyDeviceToHost);
        auto end_raycast = chrono::steady_clock::now();



        auto end_total = chrono::steady_clock::now();
        cout << "filter (ms): " << chrono::duration_cast<chrono::milliseconds>(end_filt-start_filt).count() << endl;
        cout << "tsdf (ms): " << chrono::duration_cast<chrono::milliseconds>(end_tsdf-start_tsdf).count() << endl;
        cout << "raycast (ms): " << chrono::duration_cast<chrono::milliseconds>(end_raycast-start_raycast).count() << endl;
        cout << "total (ms): " << chrono::duration_cast<chrono::milliseconds>(end_total-start_total).count() << endl;
        cout << "-----" << endl;
        //-------------------------- plot results ------------------------
        if (PLOT_RECONSTRUCTION) {
            float cpp_normals[numVertices][3];
            for (unsigned int i = 0; i < numVertices; i++) {
                cpp_normals[i][0] = normals[i].val[0];
                cpp_normals[i][1] = normals[i].val[1];
                cpp_normals[i][2] = normals[i].val[2];
            }

            cv::Mat normalsMap_Vis = cv::Mat(static_cast<int>(depthHeight), static_cast<int>(depthWidth), CV_32FC3, predictedNormals);
//            cv::Mat curr_normals = cv::Mat(static_cast<int>(depthHeight), static_cast<int>(depthWidth), CV_32FC3, cpp_normals);
            cv::Mat phong_mat = cv::Mat(static_cast<int>(depthHeight), static_cast<int>(depthWidth), CV_32F, phongSurface);
//            cv::Mat phong_mat_curr = cv::Mat(static_cast<int>(depthHeight), static_cast<int>(depthWidth), CV_32F, phongSurface_curr);

            cv::imshow("Normal Map", normalsMap_Vis);
//            cv::imshow("Curr normals", curr_normals);
            cv::imshow("Phong Shading ", phong_mat);
//            cv::imshow("Phong Surface curr ", phong_mat_curr);
//            cv::imshow("Filtered Depth Map ", filt_depth_mat);
//            cv::imshow("Initial Depth ", depth_mat);
            waitKey(10);
        }
        if (PLOT_DEPTH_LEVELS){
            frame.plotDepthAndNormals();
        }
    }
    return 0;
}